#include <stdio.h>

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>



__global__ void
copy_kernel(const hipFloatComplex* in, hipFloatComplex* out, int batch_size, int load = 1)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int n = batch_size;
    
    if (i < n) 
    {
        for (int x = 0; x < load; x++) 
        {
            out[i].x = in[i].x;
            out[i].y = in[i].y;
        }
    }
}

void apply_copy(const hipFloatComplex* in,
                hipFloatComplex* out,
                int grid_size,
                int block_size,
                int load,
                hipStream_t stream)
{
    int batch_size = block_size * grid_size;
    
    copy_kernel<<<grid_size, block_size, 0, stream>>>(in, out, batch_size, load);
    
#if 1
    hipError_t cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess)
    {
        fprintf(stderr, "ERROR: kernel launch failed - \"%s\" (%d).\n", 
                hipGetErrorString(cudaerr), int(cudaerr));
    }
#endif
}

void get_block_and_grid(int* minGrid, int* minBlock)
{
    hipError_t rc = hipOccupancyMaxPotentialBlockSize(minGrid, minBlock, copy_kernel, 0, 0);
    if (rc != hipSuccess)
    {
        fprintf(stderr, "ERROR: - \"%s\" (%d).\n", 
                hipGetErrorString(rc), int(rc));
    }
}
